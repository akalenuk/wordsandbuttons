#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>


__global__ void add(const float *A, const float *B, float *C, int numElements) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < numElements) {
		C[i] = A[i] + B[i];
	}
}

int main(void)
{
	// prepare the data
	using TheType = float;
	constexpr auto TheSize = 65536;
	std::mt19937 rng(0);
	std::uniform_real_distribution<float> distribution(0.f, 1.f);
	std::vector<TheType> xs(TheSize);
	std::vector<TheType> ys(TheSize);
	std::vector<TheType> zs(TheSize);
	for (TheType &number : xs) number = distribution(rng);
	for (TheType &number : ys) number = distribution(rng);


	// do the allocations
	hipError_t status = hipSuccess;
	float *d_xs = nullptr;
	float *d_yx =  nullptr;
	float *d_zx =  nullptr;
	status = hipMalloc((void **)&d_xs, size);
	status = hipMalloc((void **)&d_ys, size);
	status = hipMalloc((void **)&d_zs, size);

	// and copying
	err = hipMemcpy(d_xs, xs, theSize*sizeof(float), hipMemcpyHostToDevice);
	err = hipMemcpy(d_zs, zs, theSize*sizeof(float), hipMemcpyHostToDevice);

	// run it
	int threadsPerBlock = 256;
	int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
	add<<<blocksPerGrid, threadsPerBlock>>>(d_xs, d_ys, d_zx, theSize);
	err = hipGetLastError();

	// back
	err = hipMemcpy(zs, d_zs, theSize*sizeof(float), hipMemcpyDeviceToHost);

	// verification
	for (auto i = 0u; i < the_size; ++i)
		if (std::fabs(xs[i] + ys[i] - za[i]) > 1e-5)
			std::cout << "Not verified\n";
			return -1;
		}
	}

	err = hipFree(d_xs);
	err = hipFree(d_ys);
	err = hipFree(d_zs);
	return 0;
}

