#include <iostream>
#include <vector>
#include <random>
#include <hip/hip_runtime.h>


using TheType = float;
constexpr auto TheSize = 65536u*128u;
constexpr auto TheSizeInBytes = TheSize*sizeof(TheType);
constexpr auto TheInnerLoop = 256u;
	
__global__ void add(const float *xs1, const float *xs2, float *ys, int size) {
	int i = (blockDim.x * blockIdx.x + threadIdx.x);
	auto res = 0.f;
	for(auto j = 0u; j < TheInnerLoop; ++j) {
		if (i < size) {
			res += xs1[i+j] + xs2[i+j];
		}
	}
	ys[i] = res;
}

__global__ void mul(const float *xs1, const float *xs2, float *ys, int size) {
	int i = (blockDim.x * blockIdx.x + threadIdx.x);
	auto res = 0.f;
	for(auto j = 0u; j < TheInnerLoop; ++j) {
		if (i < size) {
			res += xs1[i+j] * xs2[i+j];
		}
	}
	ys[i] = res;
}

__global__ void div(const float *xs1, const float *xs2, float *ys, int size) {
	int i = (blockDim.x * blockIdx.x + threadIdx.x);
	auto res = 0.f;
	for(auto j = 0u; j < TheInnerLoop; ++j) {
		if (i < size) {
			res += xs1[i+j] / xs2[i+j];
		}
	}
	ys[i] = res;
}

__global__ void std_sin(const float *xs1, const float *xs2, float *ys, int size) {
	int i = (blockDim.x * blockIdx.x + threadIdx.x);
	auto res = 0.f;
	for(auto j = 0u; j < TheInnerLoop; ++j) {
		if (i < size) {
			res += std::sin(xs1[i+j]);
		}
	}
	ys[i] = res;
}

__global__ void poly_sin(const float *xs1, const float *xs2, float *ys, int size) {
	int i = (blockDim.x * blockIdx.x + threadIdx.x);
	auto res = 0.f;
	for(auto j = 0u; j < TheInnerLoop; ++j) {
		if (i < size) {
			const auto x = xs1[i+j];
			res += -0.000182690409228785*x*x*x*x*x*x*x+0.00830460224186793*x*x*x*x*x+-0.166651012143690*x*x*x+x;
		}
	}
	ys[i] = res;
}


#define attempt(smth) {auto s=(smth);if(s!=hipSuccess){std::cout << hipGetErrorString(s) << " at " << __LINE__ << "\n"; return -1;}}

#define measure(smth) {\
	/*timestamp start*/\
	hipEvent_t start;\
	hipEventCreate(&start);\
	hipEventRecord(start, 0);\
	hipEvent_t stop;\
	hipEventCreate(&stop); /*here so it wouldn't interfere with the measurement*/\
\
	/* run it*/\
	int threadsPerBlock = 256;\
	int blocksPerGrid = (TheSize - TheInnerLoop + threadsPerBlock - 1) / threadsPerBlock;\
	smth<<<blocksPerGrid, threadsPerBlock>>>(d_xs, d_ys, d_zs, TheSize);\
	attempt(hipGetLastError());\
	attempt(hipDeviceSynchronize());\
\
	/* timestamp stop*/\
	hipEventRecord(stop, 0); \
	hipEventSynchronize(stop);\
	float elapsedTime;\
	hipEventElapsedTime(&elapsedTime, start, stop);\
	std::cout << "Time of " << #smth << ": " << elapsedTime << "\n";}\


int main(void)
{
	// prepare the data
	std::mt19937 rng(0);
	std::uniform_real_distribution<TheType> distribution(0.f, 1.f);
	std::vector<TheType> xs(TheSize);
	std::vector<TheType> ys(TheSize);
	std::vector<TheType> zs(TheSize);
	for (TheType &number : xs) number = distribution(rng);
	for (TheType &number : ys) number = distribution(rng);


	// do the allocations
	float *d_xs = nullptr;
	float *d_ys = nullptr;
	float *d_zs = nullptr;
	attempt(hipMalloc((void **)&d_xs, TheSizeInBytes));
	attempt(hipMalloc((void **)&d_ys, TheSizeInBytes));
	attempt(hipMalloc((void **)&d_zs, TheSizeInBytes));

	// and copying
	attempt(hipMemcpy(d_xs, xs.data(), TheSizeInBytes, hipMemcpyHostToDevice));
	attempt(hipMemcpy(d_ys, ys.data(), TheSizeInBytes, hipMemcpyHostToDevice));

	measure(add);
	measure(mul);
	measure(div);
	measure(std_sin);
	measure(poly_sin);

	// back (for debug, don't really want it)
	attempt(hipMemcpy(zs.data(), d_zs, TheSizeInBytes, hipMemcpyDeviceToHost));

	attempt(hipFree(d_xs));
	attempt(hipFree(d_ys));
	attempt(hipFree(d_zs));
	return 0;
}

