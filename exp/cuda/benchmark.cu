#include <iostream>
#include <vector>
#include <random>
#include <hip/hip_runtime.h>


__global__ void add(const float *xs1, const float *xs2, float *ys, int size) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < size) {
		ys[i] = xs1[i] + xs2[i];
	}
}

#define attempt(smth) {auto s=(smth);if(s!=hipSuccess){std::cout << hipGetErrorString(s) << "\n"; return -1;}}

#define measure(smth) {\
	/*timestamp start*/\
	hipEvent_t start;\
	hipEventCreate(&start);\
	hipEventRecord(start, 0);\
	hipEvent_t stop;\
	hipEventCreate(&stop); /*here so it wouldn't interfere with the measurement*/\
\
	/* run it*/\
	int threadsPerBlock = 256;\
	int blocksPerGrid = (TheSize + threadsPerBlock - 1) / threadsPerBlock;\
	smth<<<blocksPerGrid, threadsPerBlock>>>(d_xs, d_ys, d_zs, TheSize);\
	attempt(hipGetLastError());\
	attempt(hipDeviceSynchronize());\
\
	/* timestamp stop*/\
	hipEventRecord(stop, 0); \
	hipEventSynchronize(stop);\
	float elapsedTime;\
	hipEventElapsedTime(&elapsedTime, start, stop);\
	std::cout << "Time of " << #smth << ": " << elapsedTime << "\n";}\


int main(void)
{
	// prepare the data
	using TheType = float;
	constexpr auto TheSize = 65536u*128u;
	constexpr auto TheSizeInBytes = TheSize*sizeof(TheType);
	std::mt19937 rng(0);
	std::uniform_real_distribution<TheType> distribution(0.f, 1.f);
	std::vector<TheType> xs(TheSize);
	std::vector<TheType> ys(TheSize);
	std::vector<TheType> zs(TheSize);
	for (TheType &number : xs) number = distribution(rng);
	for (TheType &number : ys) number = distribution(rng);


	// do the allocations
	float *d_xs = nullptr;
	float *d_ys = nullptr;
	float *d_zs = nullptr;
	attempt(hipMalloc((void **)&d_xs, TheSizeInBytes));
	attempt(hipMalloc((void **)&d_ys, TheSizeInBytes));
	attempt(hipMalloc((void **)&d_zs, TheSizeInBytes));

	// and copying
	attempt(hipMemcpy(d_xs, xs.data(), TheSizeInBytes, hipMemcpyHostToDevice));
	attempt(hipMemcpy(d_ys, ys.data(), TheSizeInBytes, hipMemcpyHostToDevice));

	measure(add);

	// back (for debug, don't really want it)
	attempt(hipMemcpy(zs.data(), d_zs, TheSizeInBytes, hipMemcpyDeviceToHost));

	attempt(hipFree(d_xs));
	attempt(hipFree(d_ys));
	attempt(hipFree(d_zs));
	return 0;
}

