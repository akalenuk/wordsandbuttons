#include <iostream>
#include <vector>
#include <random>
#include <hip/hip_runtime.h>


using TheType = float;
constexpr auto TheSize = 65536u*128u;
constexpr auto TheSizeInBytes = TheSize*sizeof(TheType);
constexpr auto TheInnerLoop = 256u;
	
__global__ void add(const float *xs1, const float *xs2, float *ys, int size) {
	int i = (blockDim.x * blockIdx.x + threadIdx.x);
	auto res = 0.f;
	for(auto j = 0u; j < TheInnerLoop; ++j) {
		res += xs1[i+j] + xs2[i+j];
	}
	ys[i] = res;
}

__global__ void mul(const float *xs1, const float *xs2, float *ys, int size) {
	int i = (blockDim.x * blockIdx.x + threadIdx.x);
	auto res = 0.f;
	for(auto j = 0u; j < TheInnerLoop; ++j) {
		res += xs1[i+j] * xs2[i+j];
	}
	ys[i] = res;
}

__global__ void div(const float *xs1, const float *xs2, float *ys, int size) {
	int i = (blockDim.x * blockIdx.x + threadIdx.x);
	auto res = 0.f;
	for(auto j = 0u; j < TheInnerLoop; ++j) {
		res += xs1[i+j] / xs2[i+j];
	}
	ys[i] = res;
}

__global__ void std_sqrt(const float *xs1, const float *xs2, float *ys, int size) {
	int i = (blockDim.x * blockIdx.x + threadIdx.x);
	auto res = 0.f;
	for(auto j = 0u; j < TheInnerLoop; ++j) {
		res += std::sqrt(xs1[i+j]);
	}
	ys[i] = res;
}

__global__ void std_sin(const float *xs1, const float *xs2, float *ys, int size) {
	int i = (blockDim.x * blockIdx.x + threadIdx.x);
	auto res = 0.f;
	for(auto j = 0u; j < TheInnerLoop; ++j) {
		res += std::sin(xs1[i+j]);
	}
	ys[i] = res;
}

__global__ void poly_sin(const float *xs1, const float *xs2, float *ys, int size) {
	int i = (blockDim.x * blockIdx.x + threadIdx.x);
	auto res = 0.f;
	for(auto j = 0u; j < TheInnerLoop; ++j) {
		const auto x = xs1[i+j];
		res += -0.000182690409228785*x*x*x*x*x*x*x+0.00830460224186793*x*x*x*x*x-0.166651012143690*x*x*x+x;
	}
	ys[i] = res;
}

__global__ void poly_sin2(const float *xs1, const float *xs2, float *ys, int size) {
	int i = (blockDim.x * blockIdx.x + threadIdx.x);
	auto res = 0.f;
	for(auto j = 0u; j < TheInnerLoop; ++j) {
		const auto x = xs1[i+j];
		res += x*x*x*(x*x*(-0.000182690409228785*x*x+0.00830460224186793)-0.166651012143690)+x;
	}
	ys[i] = res;
}

__global__ void poly_sin3(const float *xs1, const float *xs2, float *ys, int size) {
	int i = (blockDim.x * blockIdx.x + threadIdx.x);
	auto res = 0.f;
	for(auto j = 0u; j < TheInnerLoop; ++j) {
		const auto x = xs1[i+j];
		res += -0.000182690409228785f*x*x*x*x*x*x*x+0.00830460224186793f*x*x*x*x*x-0.166651012143690f*x*x*x+x;
	}
	ys[i] = res;
}

__global__ void poly_sin4(const float *xs1, const float *xs2, float *ys, int size) {
	int i = (blockDim.x * blockIdx.x + threadIdx.x);
	auto res = 0.f;
	for(auto j = 0u; j < TheInnerLoop; ++j) {
		const auto x = xs1[i+j];
		res += x*x*x*(x*x*(-0.000182690409228785f*x*x+0.00830460224186793f)-0.166651012143690f)+x;
	}
	ys[i] = res;
}

__global__ void logical_and(const float *xs1, const float *xs2, float *ys, int size) {
	int i = (blockDim.x * blockIdx.x + threadIdx.x);
	bool all_gt = true;
	for(auto j = 0u; j < TheInnerLoop - 3; ++j) {
		all_gt = all_gt && (xs1[i+j] > xs1[i+j]) && (xs1[i+j+1] > xs1[i+j+1]) && (xs1[i+j+2] > xs1[i+j]+2) && (xs1[i+j+3] > xs1[i+j+3]);
	}
	ys[i] = all_gt ? 1.f : 0.f;
}

__global__ void bit_and(const float *xs1, const float *xs2, float *ys, int size) {
	int i = (blockDim.x * blockIdx.x + threadIdx.x);
	bool all_gt = true;
	for(auto j = 0u; j < TheInnerLoop - 3; ++j) {
		all_gt = all_gt & (xs1[i+j] > xs1[i+j]) & (xs1[i+j+1] > xs1[i+j+1]) & (xs1[i+j+2] > xs1[i+j]+2) & (xs1[i+j+3] > xs1[i+j+3]);
	}
	ys[i] = all_gt ? 1.f : 0.f;
}

__global__ void mul_and(const float *xs1, const float *xs2, float *ys, int size) {
	int i = (blockDim.x * blockIdx.x + threadIdx.x);
	bool all_gt = true;
	for(auto j = 0u; j < TheInnerLoop - 3; ++j) {
		all_gt = all_gt * (xs1[i+j] > xs1[i+j]) * (xs1[i+j+1] > xs1[i+j+1]) * (xs1[i+j+2] > xs1[i+j]+2) * (xs1[i+j+3] > xs1[i+j+3]);
	}
	ys[i] = all_gt ? 1.f : 0.f;
}

#define swap(a, b) {auto c = a; a = b; b = c;}

__global__ void sort(const float *xs1, const float *xs2, float *ys, int size) {
	int i = (blockDim.x * blockIdx.x + threadIdx.x);
	float checksum = 0.;
	for(auto j = 0u; j < TheInnerLoop - 2; ++j) {
		double s[3] = {xs1[i+j], xs1[i+j+1], xs1[i+j+2]};
		if(s[0] > s[1])
			swap(s[0], s[1]);
		if(s[1] > s[2])
			swap(s[1], s[2]);
		if(s[0] > s[1])
			swap(s[0], s[1]);
		checksum += s[0] + 2*s[1] + 3*s[3];
	}
	ys[i] = checksum;
}

__global__ void nano_sort(const float *xs1, const float *xs2, float *ys, int size) {
	int i = (blockDim.x * blockIdx.x + threadIdx.x);
	float checksum = 0.;
	for(auto j = 0u; j < TheInnerLoop - 2; ++j) {
		double sortable[3] = {xs1[i+j], xs1[i+j+1], xs1[i+j+2]};
		const auto a = sortable[0];
		const auto b = sortable[1];
		const auto c = sortable[2];
		sortable[int(a > b) + int(a > c)] = a;
		sortable[int(b >= a) + int(b > c)] = b;
		sortable[int(c >= a) + int(c >= b)] = c;
		checksum += sortable[0] + 2*sortable[1] + 3*sortable[3];
	}
	ys[i] = checksum;
}

#define attempt(smth) {auto s=(smth);if(s!=hipSuccess){std::cout << hipGetErrorString(s) << " at " << __LINE__ << "\n"; return -1;}}

#define measure(smth) {\
	/*timestamp start*/\
	hipEvent_t start;\
	hipEventCreate(&start);\
	hipEventRecord(start, 0);\
	hipEvent_t stop;\
	hipEventCreate(&stop); /*here so it wouldn't interfere with the measurement*/\
\
	/* run it*/\
	int threadsPerBlock = 256;\
	int blocksPerGrid = (TheSize - TheInnerLoop + threadsPerBlock - 1) / threadsPerBlock;\
	smth<<<blocksPerGrid, threadsPerBlock>>>(d_xs, d_ys, d_zs, TheSize);\
	attempt(hipGetLastError());\
	attempt(hipDeviceSynchronize());\
\
	/* timestamp stop*/\
	hipEventRecord(stop, 0); \
	hipEventSynchronize(stop);\
	float elapsedTime;\
	hipEventElapsedTime(&elapsedTime, start, stop);\
	std::cout << "Time of " << #smth << ": " << elapsedTime << "\n";}\


int main(void)
{
	// prepare the data
	std::mt19937 rng(0);
	std::uniform_real_distribution<TheType> distribution(0.f, 1.f);
	std::vector<TheType> xs(TheSize);
	std::vector<TheType> ys(TheSize);
	std::vector<TheType> zs(TheSize);
	for (TheType &number : xs) number = distribution(rng);
	for (TheType &number : ys) number = distribution(rng);


	// do the allocations
	float *d_xs = nullptr;
	float *d_ys = nullptr;
	float *d_zs = nullptr;
	attempt(hipMalloc((void **)&d_xs, TheSizeInBytes));
	attempt(hipMalloc((void **)&d_ys, TheSizeInBytes));
	attempt(hipMalloc((void **)&d_zs, TheSizeInBytes));

	// and copying
	attempt(hipMemcpy(d_xs, xs.data(), TheSizeInBytes, hipMemcpyHostToDevice));
	attempt(hipMemcpy(d_ys, ys.data(), TheSizeInBytes, hipMemcpyHostToDevice));

	measure(add);
	measure(mul);
	measure(div);
	measure(std_sqrt);
	measure(std_sin);
	measure(poly_sin);
	measure(poly_sin2);
	measure(poly_sin3);
	measure(poly_sin4);
	measure(logical_and);
	measure(bit_and);
	measure(mul_and);
	measure(sort);
	measure(nano_sort);

	// back (for debug, don't really want it)
	attempt(hipMemcpy(zs.data(), d_zs, TheSizeInBytes, hipMemcpyDeviceToHost));

	attempt(hipFree(d_xs));
	attempt(hipFree(d_ys));
	attempt(hipFree(d_zs));
	return 0;
}

